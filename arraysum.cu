#include "hip/hip_runtime.h"
// CUDA Runtime
#include <hip/hip_runtime.h>

// Utilities and system includes
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>



__global__ void reduce0(double *g_idata, double *g_odata, unsigned int n)
{
	__shared__ double sdata[];

	// load shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

	sdata[tid] = (i < n) ? g_idata[i] : 0;

	__syncthreads();

	// do reduction in shared mem
	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
		// modulo arithmetic is slow!
		if ((tid % (2 * s)) == 0)
		{
			sdata[tid] += sdata[tid + s];
		}

		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

hipError_t reduceWithCuda(double *hdata, int size)
{

}

void cleanup(hipError_t status)
{
	hipDeviceReset();
	exit(status == hipSuccess ? 0 : 1);
}

int main(int argc, char **argv)
{
	int size = 1024;
	int bytes = size * sizeof(double);
	int threads = 1024;
	int blocks = 1;
	double *hostData = (double *)malloc(bytes);
	for (int i = 0; i<size; i++)
		hostData[i] = (rand() & 0xFF) / (double)RAND_MAX;
	
	hipError_t exitcode;

	exitcode = hipSetDevice(0);
	if (exitcode != hipSuccess)
	{
		printf("Error setting device 0: %d\n", exitcode);
		cleanup(exitcode);
	}
	
	double *devInputData;
	exitcode = hipMalloc(&devInputData, bytes);
	if (exitcode != cudaSucces)
	{
		printf("Error in hipMalloc %d\n", exitcode);
		cleanup(exitcode);
	}

	double *devOutputData;
	exitcode = hipMalloc(&devOutputData, sizeof(double));
	if (exitcode != cudaSucces)
	{
		printf("Error in hipMalloc for output %d\n", exitcode);
		cleanup(exitcode);
	}

	exitcode = hipMemcpy(devInputData, hostData, bytes, hipMemcpyHostToDevice);
	if (exitcode != cudaSucces)
	{
		printf("Error in hipMemcpy to device %d\n", exitcode);
		cleanup(exitcode);
	}

	dim3 dimBlock(threads, 1, 1);
	dim3 dimGrid(blocks, 1, 1);

	reduce0 << <dimGrid, dimBlock >> >(devInputData, devOutputData, bytes);

	exitcode = hipGetLastError();
	if (exitcode != hipSuccess) {
		fprintf(stderr, "reduce0 launch failed: %s\n", hipGetErrorString(exitcode));
		cleanup(exitcode);
	}

	exitcode = hipDeviceSynchronize();
	if (exitcode != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", exitcode);
		cleanup(exitcode);
	}

	cleanup(hipSuccess);
}