#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <ctime>
#include <time.h>
#include <sstream>
#include <string>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

//Kernel
__global__ void reduce0(int *g_idata, int *g_odata, int size) {
	
	//Tablica wartości do zsumowania w pamięci współdzielonej (w obrębie bloku)
	extern __shared__ int sdata[];
	
	//Pobierz id wątku (w bloku)
	unsigned int tid = threadIdx.x;
	//Oblicz globalny id wątku (we wszystkich blokach)
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	//Wyzeru pozycję w tablicy odpowiadającej wątkowi
	sdata[tid] = 0;
	if(i < size)
		//Pobierz wartość do pamięci współdzielonej
		sdata[tid] = g_idata[i];
	//Synchronizacja (aby wszystkie wątki zdążyły pobrać dane)
	__syncthreads();
	
	//Sumowanie
	for(unsigned int s=1; s < blockDim.x; s *= 2) {
		//Jeżeli id wątku jest parzysty ...
        if (tid % (2*s) == 0) {
			//... zlicz wartość swoją i nieparzystego sąsiada
			sdata[tid] += sdata[tid + s];
        }
		//Synchronizacja (aby wszystkie wątki zdążyły posumować)
        __syncthreads();
    }
	
	//Zapisz rezultat z tego bloku do pamięci
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main(void) {
	//Deklaracja rozmiaru
	int size = 939289;
	//Wektor wejściowy hosta
	thrust::host_vector<int> data_h_i(size, 1);

	//Liczba wątków na blok
	int threadsPerBlock = 1024;
	//Liczba bloków (na początku)
	int totalBlocks = (size+(threadsPerBlock-1))/threadsPerBlock;

	//Wektor wejściowy i wyjściowy device
	thrust::device_vector<int> data_v_i = data_h_i;
	thrust::device_vector<int> data_v_o(totalBlocks);

	//Wskaźniki na wektory device
	int* output = thrust::raw_pointer_cast(data_v_o.data());
	int* input = thrust::raw_pointer_cast(data_v_i.data());
	
	//Czy kolej na zamianę tablicy wejściowej z wyjściową?
	bool turn = true;
	
	hipError_t error;
	
	hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	
	while(true) {	
		if(turn) {
			//Odpal kernel (tablica wejściowa jako input, wyjściowa jako output
			reduce0<<<totalBlocks, threadsPerBlock, threadsPerBlock*sizeof(int)>>>(input, output, size);
			turn = false;
		} else {
			//Odpal kernel (tablica wyjściowa jako input, wejściowa jako output
			reduce0<<<totalBlocks, threadsPerBlock, threadsPerBlock*sizeof(int)>>>(output, input, size);
			turn = true;
		}
		
		//Jeżeli został jeden blok, to obliczenia zostały zakończone
		if(totalBlocks == 1) break;
		
		//Korzystaj tylko z zakresu tablicy odpowiadającemu liczbie bloków z poprzedniej iteracji
		size = totalBlocks;
		//Oblicz nową liczbę bloków
		totalBlocks = ceil((double)totalBlocks/threadsPerBlock);
	}
	
	error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	
	//Wektor wyjściowy hosta
	thrust::host_vector<int> data_h_o;
	  
	//Pobierz wynik
	if(turn)
		//Wynik w tablicy wejściowej device
		data_h_o = data_v_i;
	else
		//Wynik w tablicy wyjściowej device
		data_h_o = data_v_o;
	
	//Wyczyść wektory
	data_v_i.clear();
	data_v_i.shrink_to_fit();
	  
	data_v_o.clear();
	data_v_o.shrink_to_fit();
	  
	//Wypisz wynik
	cout<< "Wynik: " << data_h_o[0] << endl << "W czasie:" << msecTotal << endl;

	return 0;
}
