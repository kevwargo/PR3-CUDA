#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <ctime>
#include <time.h>
#include <sstream>
#include <string>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

//Kernel
__global__ void reduce0(int *g_idata, int *g_odata, int size) {
	
	//Tablica warto�ci do zsumowania w pami�ci wsp�dzielonej (w obr�bie bloku)
	extern __shared__ int sdata[];
	
	//Pobierz id w�tku (w bloku)
	unsigned int tid = threadIdx.x;
	//Oblicz globalny id w�tku (we wszystkich blokach)
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	//Wyzeru pozycj� w tablicy odpowiadaj�cej w�tkowi
	sdata[tid] = 0;
	if(i < size)
		//Pobierz warto�� do pami�ci wsp�dzielonej
		sdata[tid] = g_idata[i];
	//Synchronizacja (aby wszystkie w�tki zd��y�y pobra� dane)
	__syncthreads();
	
	//Sumowanie
	for(unsigned int s=1; s < blockDim.x; s *= 2) {
		//Je�eli id w�tku jest parzysty ...
        if (tid % (2*s) == 0) {
			//... zlicz warto�� swoj� i nieparzystego s�siada
			sdata[tid] += sdata[tid + s];
        }
		//Synchronizacja (aby wszystkie w�tki zd��y�y posumowa�)
        __syncthreads();
    }
	
	//Zapisz rezultat z tego bloku do pami�ci
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main(void) {
	//Deklaracja rozmiaru
	int size = 939289;
	//Wektor wej�ciowy hosta
	thrust::host_vector<int> data_h_i(size, 1);

	//Liczba w�tk�w na blok
	int threadsPerBlock = 1024;
	//Liczba blok�w (na pocz�tku)
	int totalBlocks = (size+(threadsPerBlock-1))/threadsPerBlock;

	//Wektor wej�ciowy i wyj�ciowy device
	thrust::device_vector<int> data_v_i = data_h_i;
	thrust::device_vector<int> data_v_o(totalBlocks);

	//Wska�niki na wektory device
	int* output = thrust::raw_pointer_cast(data_v_o.data());
	int* input = thrust::raw_pointer_cast(data_v_i.data());
	
	//Czy kolej na zamian� tablicy wej�ciowej z wyj�ciow�?
	bool turn = true;
	
	hipError_t error;
	
	hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	
	while(true) {	
		if(turn) {
			//Odpal kernel (tablica wej�ciowa jako input, wyj�ciowa jako output
			reduce0<<<totalBlocks, threadsPerBlock, threadsPerBlock*sizeof(int)>>>(input, output, size);
			turn = false;
		} else {
			//Odpal kernel (tablica wyj�ciowa jako input, wej�ciowa jako output
			reduce0<<<totalBlocks, threadsPerBlock, threadsPerBlock*sizeof(int)>>>(output, input, size);
			turn = true;
		}
		
		//Je�eli zosta� jeden blok, to obliczenia zosta�y zako�czone
		if(totalBlocks == 1) break;
		
		//Korzystaj tylko z zakresu tablicy odpowiadaj�cemu liczbie blok�w z poprzedniej iteracji
		size = totalBlocks;
		//Oblicz now� liczb� blok�w
		totalBlocks = ceil((double)totalBlocks/threadsPerBlock);
	}
	
	error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	
	//Wektor wyj�ciowy hosta
	thrust::host_vector<int> data_h_o;
	  
	//Pobierz wynik
	if(turn)
		//Wynik w tablicy wej�ciowej device
		data_h_o = data_v_i;
	else
		//Wynik w tablicy wyj�ciowej device
		data_h_o = data_v_o;
	
	//Wyczy�� wektory
	data_v_i.clear();
	data_v_i.shrink_to_fit();
	  
	data_v_o.clear();
	data_v_o.shrink_to_fit();
	  
	//Wypisz wynik
	cout<< "Wynik: " << data_h_o[0] << endl << "W czasie:" << msecTotal << endl;

	return 0;
}
