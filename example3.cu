#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <ctime>
#include <time.h>
#include <sstream>
#include <string>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;


__global__ void reduce0(int *g_idata, int *g_odata, int size){

   extern __shared__ int sdata[];

   unsigned int tid = threadIdx.x;
   unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
   sdata[tid] = 0;
   if(i<size)
     sdata[tid] = g_idata[i];
   __syncthreads();

	for(unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

   if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main(void){

  int size = 939289;
  thrust::host_vector<int> data_h_i(size, 1);

  //initialize the data, all values will be 1
  //so the final sum will be equal to size

  int threadsPerBlock = 1024;
  int totalBlocks = (size+(threadsPerBlock-1))/threadsPerBlock;
  
  thrust::device_vector<int> data_v_i = data_h_i;
  thrust::device_vector<int> data_v_o(totalBlocks);

  int* output = thrust::raw_pointer_cast(data_v_o.data());
  int* input = thrust::raw_pointer_cast(data_v_i.data());
  
  bool turn = true;
  
	hipError_t error;
	
	hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	
	while(true) {	
		if(turn) {
			//Odpal kernel (tablica wej�ciowa jako input, wyj�ciowa jako output
			reduce0<<<totalBlocks, threadsPerBlock, threadsPerBlock*sizeof(int)>>>(input, output, size);
			turn = false;
		} else {
			//Odpal kernel (tablica wyj�ciowa jako input, wej�ciowa jako output
			reduce0<<<totalBlocks, threadsPerBlock, threadsPerBlock*sizeof(int)>>>(output, input, size);
			turn = true;
		}
		
		//Je�eli zosta� jeden blok, to obliczenia zosta�y zako�czone
		if(totalBlocks == 1) break;
		
		//Korzystaj tylko z zakresu tablicy odpowiadaj�cemu liczbie blok�w z poprzedniej iteracji
		size = totalBlocks;
		//Oblicz now� liczb� blok�w
		totalBlocks = ceil((double)totalBlocks/threadsPerBlock);
	}
	
	error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	
	//Wektor wyj�ciowy hosta
	thrust::host_vector<int> data_h_o;
	  
	//Pobierz wynik
	if(turn)
		//Wynik w tablicy wej�ciowej device
		data_h_o = data_v_i;
	else
		//Wynik w tablicy wyj�ciowej device
		data_h_o = data_v_o;
	
	//Wyczy�� wektory
	data_v_i.clear();
	data_v_i.shrink_to_fit();
	  
	data_v_o.clear();
	data_v_o.shrink_to_fit();
	  
	//Wypisz wynik
	cout<< "Wynik: " << data_h_o[0] << endl << "W czasie:" << msecTotal << endl;

	return 0;
}